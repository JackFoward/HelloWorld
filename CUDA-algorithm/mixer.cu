#include "hip/hip_runtime.h"
/*
 *  Copyright (C) 2017 RidgeRun, LLC (http://www.ridgerun.com)
 *  All Rights Reserved.
 *  Author: Jennifer Caballero <jennifer.caballero@ridgerun.com>
 *
 *  The contents of this software are proprietary and confidential to RidgeRun,
 *  LLC.  No part of this program may be photocopied, reproduced or translated
 *  into another programming language without prior written consent of
 *  RidgeRun, LLC.  The user is free to modify the source code after obtaining
 *  a software license from RidgeRun.  All source code changes must be provided    *  back to RidgeRun without any encumbrance.
 */
//hi there
#include "stdio.h"
#include "mixer.h"

__global__ void
cudaImageMixer (unsigned char *grayIn, unsigned char *colorInY, unsigned char *colorInU,
    unsigned char *colorInV, unsigned char *colorOut, int width, int height,
      unsigned char threshold, unsigned char yColor, unsigned char uColor, unsigned char vColor, int pitch)
{
  /* Compute indexes */
  int yIndex = (2 * width * blockIdx.y) + (2 * threadIdx.x) +
      ((width / 2) * (blockIdx.x));

  int yIndexPitchOdd = yIndex + (2 * pitch * blockIdx.y);
  int yIndexpitchEven = yIndexPitchOdd + pitch;

  int uIndex = (width * height) + ((width / 2) * blockIdx.y) +
      threadIdx.x + ((width / 4) * blockIdx.x);

  int vIndex = uIndex + ((width * height) / 4);
  int uvIndexPitch = uIndex + ((pitch/2) * blockIdx.y) - (width * height);


  /* Test threshold and assign U and V accordingly */

  if (grayIn[yIndexPitchOdd] > threshold || grayIn[yIndexPitchOdd + 1] > threshold ||
      grayIn[yIndexpitchEven + width] > threshold
      || grayIn[yIndexpitchEven + width + 1] > threshold) {
    colorOut[uIndex] = uColor;
    colorOut[vIndex] = vColor;
  } else {
    colorOut[uIndex] = colorInU[uvIndexPitch];
    colorOut[vIndex] = colorInV[uvIndexPitch];
  }

  colorOut[yIndex] = colorInY[yIndexPitchOdd];
  colorOut[yIndex + 1] = colorInY[yIndexPitchOdd + 1];
  colorOut[yIndex + width] = colorInY[yIndexpitchEven + width];
  colorOut[yIndex + width + 1] = colorInY[yIndexpitchEven + width + 1];

}

bool
imageMixer (unsigned char *grayIn, unsigned char *colorInY, unsigned char *colorInU,
    unsigned char *colorInV, unsigned char *colorOut, int width, int height,
      unsigned char threshold, unsigned char *color, int pitch)
{

  hipError_t cudaErr;

  if (color == NULL) {
    printf ("Error: Please provide a color\n");
    return false;
  } else if (grayIn == NULL || colorInY == NULL || colorInU == NULL
      || colorInV == NULL || colorOut == NULL) {
    printf ("Error: NULL memory pointer in %s\n", __FUNCTION__);
    return false;
  }

  cudaImageMixer <<< dim3 (2, (height / 2), 1), dim3 (width / 4, 1,
      1) >>> (grayIn, colorInY, colorInU, colorInV, colorOut, width, height, threshold, color[0],
      color[1], color[2], pitch);

  cudaErr = hipGetLastError ();

  if (hipSuccess != cudaErr) {
    printf ("CUDA kernel Error\n");
    return false;
  }

  hipDeviceSynchronize ();

  cudaErr = hipGetLastError ();

  if (hipSuccess != cudaErr) {
    printf ("CUDA sync Error\n");
    return false;
  }

  return true;
}

bool
imageMixerAllocateMemory (unsigned char **grayIn, unsigned char **colorInY,
    unsigned char **colorInU, unsigned char **colorInV,
      unsigned char **colorOut, int width, int height, int pitch)
{

  hipError_t cudaErr;

  if (grayIn == NULL || colorInY == NULL || colorInU == NULL|| colorInV == NULL
      || colorOut == NULL) {
    printf ("Error: NULL memory pointer in %s\n", __FUNCTION__);
    return false;
  }

  /* TX1 Hardware doesn't support more than 1024 threads/block */
  if (width / 4 > 1024) {
    printf ("Error: Max supported width is 4096\n");
    return false;
  }

  hipMallocManaged (colorInY, (width + pitch) * height);
  hipMallocManaged (colorInU, (width + pitch) * (height/4));
  hipMallocManaged (colorInV, (width + pitch) * (height/4));
  hipMallocManaged (colorOut, width * height * 1.5);
  hipMallocManaged (grayIn, (width + pitch) * height);

  cudaErr = hipGetLastError ();
  if (hipSuccess != cudaErr) {
    printf ("CUDA alloc Error\n");
    return false;
  }

  return true;
}

bool
imageMixerFreeMemory (unsigned char *grayIn, unsigned char *colorInY,
    unsigned char *colorInU, unsigned char *colorInV, unsigned char *colorOut)
{

  hipError_t cudaErr;

  if (grayIn == NULL || colorInY == NULL || colorInU == NULL || colorInV == NULL
      || colorOut == NULL) {
    printf ("Error: NULL memory pointer in %s\n", __FUNCTION__);
    return false;
  }

  hipFree (grayIn);
  hipFree (colorInY);
  hipFree (colorInU);
  hipFree (colorInV);
  hipFree (colorOut);
  cudaErr = hipGetLastError ();

  if (hipSuccess != cudaErr) {
    printf ("CUDA alloc Error\n");
    return false;
  }

  return true;
}
